#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

__global__ void device_reshape(Pair *old_entries, unsigned int old_size, Pair *new_entries, unsigned int new_size) {
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= old_size || old_entries[idx].key == KEY_INVALID)
		return;
	unsigned int hash = (((unsigned long long)(old_entries[idx].key) * 985463) % 2865417259) % new_size;
	unsigned int initial_hash = hash;
    while (true) {
        unsigned int prev = atomicCAS(&new_entries[hash].key, KEY_INVALID, old_entries[idx].key);
        if (prev == KEY_INVALID) {
			new_entries[hash].value = old_entries[idx].value;
            return;
		} else {
    		hash = (hash + 1) % new_size;
			if (hash == initial_hash)
				return;
		}
    }
}

__global__ void insert(Pair *entries, unsigned int table_len, unsigned int *keys, unsigned int* values, unsigned int numKeys, unsigned int *inserted_keys) {
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= numKeys)
		return;
	unsigned int hash = (((long long)(keys[idx]) * 985463) % 2865417259) % table_len;
	unsigned int initial_hash = hash;
    while (true) {
        unsigned int prev = atomicCAS(&entries[hash].key, KEY_INVALID, keys[idx]);
        if (prev == KEY_INVALID) {
			entries[hash].value = values[idx];
			atomicInc(inserted_keys, INT_MAX);
            return;
		} else if (prev == keys[idx]) {
			entries[hash].value = values[idx];
			return;
		}
        hash = (hash + 1) % table_len;
		if (hash == initial_hash)
			return;
    }
}

__global__ void get(Pair *entries, unsigned int table_len, unsigned int *keys, unsigned int numKeys, unsigned int* values) {
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= numKeys)
		return;
	unsigned int hash = (((long long)(keys[idx]) * 985463) % 2865417259) % table_len;
	unsigned int initial_hash = hash;
	while (true) {
		if (entries[hash].key == keys[idx]) {
			values[idx] = entries[hash].value;
			return;
		}
        hash = (hash + 1) % table_len;
		if (hash == initial_hash)
			return;
	}
}

/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) {
	glbGpuAllocator->_cudaMallocManaged((void **)&this->entries, sizeof(Pair) * size);
	this->inserted_keys = 0;
	this->table_len = size;
	hipMemset(this->entries, 0, size * sizeof(Pair));
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(this->entries);
}

/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	Pair *old_entries = this->entries;
	unsigned int old_size = this->table_len;
	unsigned int numBlocks = old_size / 256;
	glbGpuAllocator->_cudaMallocManaged((void **)&this->entries, sizeof(Pair) * numBucketsReshape);
	hipMemset(this->entries, 0, numBucketsReshape * sizeof(Pair));
	this->table_len = numBucketsReshape;
	if (old_size % 256 != 0)
		numBlocks++;
	device_reshape<<<numBlocks, 256>>>(old_entries, old_size, this->entries, this->table_len);
	hipDeviceSynchronize();
	glbGpuAllocator->_cudaFree(old_entries);
	return;
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	if (numKeys == 0)
		return false;
	if ((float)(inserted_keys + numKeys) / table_len >= 0.8f)
    reshape(table_len * 1.5f);
	unsigned int *values_GPU;
	unsigned int *keys_GPU;
	unsigned int *inserted_keys_GPU;
	unsigned int numBlocks = numKeys / 256;
	glbGpuAllocator->_cudaMalloc((void **)&values_GPU, sizeof(unsigned int) * numKeys);
	hipMemcpy(values_GPU, values, sizeof(unsigned int) * numKeys, hipMemcpyHostToDevice);
	glbGpuAllocator->_cudaMalloc((void **)&keys_GPU, sizeof(unsigned int) * numKeys);
	hipMemcpy(keys_GPU, keys, sizeof(unsigned int) * numKeys, hipMemcpyHostToDevice);
	glbGpuAllocator->_cudaMallocManaged((void **)&inserted_keys_GPU, sizeof(unsigned int));
	*inserted_keys_GPU = 0;
	if (numKeys % 256 != 0)
		numBlocks++;
	insert<<<numBlocks, 256>>>(this->entries, this->table_len, keys_GPU, values_GPU, numKeys, inserted_keys_GPU);
	hipDeviceSynchronize();
	this->inserted_keys += *inserted_keys_GPU;
	glbGpuAllocator->_cudaFree(values_GPU);
	glbGpuAllocator->_cudaFree(keys_GPU);
	glbGpuAllocator->_cudaFree(inserted_keys_GPU);
	return true;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	if (numKeys == 0)
		return NULL;
	unsigned int *values_GPU;
	unsigned int *keys_GPU;
	unsigned int blocks = numKeys / 256;
	unsigned int *values_RAM = (unsigned int *)malloc(sizeof(unsigned int) * numKeys);
	glbGpuAllocator->_cudaMallocManaged((void **)&values_GPU, sizeof(unsigned int) * numKeys);
	glbGpuAllocator->_cudaMalloc((void **)&keys_GPU, sizeof(unsigned int) * numKeys);
	hipMemcpy(keys_GPU, keys, sizeof(unsigned int) * numKeys, hipMemcpyHostToDevice);
	if (numKeys % 256 != 0)
		blocks++;
	get<<<blocks, 256>>>(this->entries, this->table_len, keys_GPU, numKeys, values_GPU);
	hipDeviceSynchronize();
	hipMemcpy(values_RAM, values_GPU, sizeof(unsigned int) * numKeys, hipMemcpyDeviceToHost);
	glbGpuAllocator->_cudaFree(values_GPU);
	glbGpuAllocator->_cudaFree(keys_GPU);
	return (int *)values_RAM;
}
